#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <cmath>

#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(error)); \
            hipDeviceReset(); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)


#define batch_size 32
#define sequence_length 1024
#define hidden_dim 512
#define n_head 8
#define head_dim 64
#define T_c 32
#define T_r 32
#define Bc 32
#define Br 32
#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define softmax_scale 0.125 // (1 / 64)

__global__ void multi_head_attention(float *d_q, float *d_k, float *d_v, float *d_S, float *d_P, float *d_o){

    int qkv_offset = (blockIdx.y * (n_head * sequence_length * head_dim)) + (blockIdx.x * (sequence_length * head_dim));
    int sp_offset = (blockIdx.y * (n_head * sequence_length * sequence_length)) + (blockIdx.x * (sequence_length * sequence_length));
    d_q += qkv_offset;
    d_k += qkv_offset;
    d_v += qkv_offset;
    d_o += qkv_offset;
    d_S += sp_offset;
    d_P += sp_offset;
    
    for (int w1 = 0; w1 < sequence_length; w1++){
        float tmp = 0.0;
        for (int k1 = 0; k1 < head_dim; k1++){
            tmp += d_q[threadIdx.x * head_dim + k1] * d_k[w1 * head_dim + k1];
        }
        d_S[threadIdx.x * sequence_length + w1] = tmp * softmax_scale;
    }
    __syncthreads();

    for (int h = 0; h < sequence_length; h++){
        if (h > threadIdx.x) {
            d_S[threadIdx.x * sequence_length + h] = -INFINITY;
        }
    }

    float denominator = 0.0;
    for (int l1 = 0; l1 < sequence_length; l1++){
        denominator += exp(d_S[threadIdx.x * sequence_length + l1]);
    }
    __syncthreads();

    for (int d1 = 0; d1 < sequence_length; d1++){
        d_P[threadIdx.x * sequence_length + d1] = exp(d_S[threadIdx.x * sequence_length + d1]) / denominator;
    }
    __syncthreads();

    for (int b1 = 0; b1 < head_dim; b1++){
        float tmp2 = 0.0;
        for (int v1 = 0; v1 < sequence_length; v1++){
            tmp2 += d_P[threadIdx.x * sequence_length + v1] * d_v[v1 * head_dim + b1];
        }
        d_o[threadIdx.x * head_dim + b1] = tmp2;
    }
}

__global__ void flash_attn_kernel_2(float *d_q, float *d_k, float *d_v, float *d_o, float *d_l, float *d_m){
    // extern __shared__ float smem[];
    // float *q_shared = smem;
    // float *k_shared = q_shared + Br * head_dim;
    // float *v_shared = k_shared + Bc * head_dim;
    // float *s_shared = v_shared + Bc * head_dim;
    // float *o_shared = s_shared + Br * Bc;
    // float *l_shared = o_shared + Br * head_dim;
    // float *m_shared = l_shared + Br;

    // extern __shared__ float smem[];
    // float *q_shared = smem;                           // Size: Br * head_dim
    // float *k_shared = q_shared + Br * head_dim;      // Size: Bc * head_dim  
    // float *v_shared = k_shared + Bc * head_dim;      // Size: Bc * head_dim
    // float *s_shared = v_shared + Bc * head_dim;      // Size: Br * Bc (FIXED!)
    // float *o_shared = s_shared + Br * Bc;            // Size: Br * head_dim (FIXED!)
    // float *l_shared = o_shared + Br * head_dim;      // Size: Br (FIXED!)
    // float *m_shared = l_shared + Br;   
    __shared__ float k_shared[Bc * head_dim];
    __shared__ float v_shared[Bc * head_dim];
    __shared__ float q_shared[Br * head_dim];
    __shared__ float o_shared[Br * head_dim];
    __shared__ float l_shared[Br];
    __shared__ float m_shared[Br];
    __shared__ float s_shared[Br * Bc];

    int q_offset = (blockIdx.y * (sequence_length * head_dim)) + (blockIdx.x * (Br * head_dim));
    int kv_offset = (blockIdx.y * (sequence_length * head_dim));
    int lm_offset = (blockIdx.y * (sequence_length)) + (blockIdx.x * (Br));

    float *d_q_orig = d_q + q_offset;
    float *d_k_orig = d_k + kv_offset;
    float *d_v_orig = d_v + kv_offset;
    float *d_o_orig = d_o + q_offset;
    float *d_l_orig = d_l + lm_offset;
    float *d_m_orig = d_m + lm_offset;

    d_q = d_q_orig;
    d_k = d_k_orig;
    d_v = d_v_orig;
    d_o = d_o_orig;
    d_l = d_l_orig;
    d_m = d_m_orig;

    for (int a1 = 0; a1 < head_dim; a1++){
        if (threadIdx.x < Bc) {
            q_shared[threadIdx.x * head_dim + a1] = d_q[threadIdx.x * head_dim + a1]; 
            o_shared[threadIdx.x * head_dim + a1] = d_o[threadIdx.x * head_dim + a1];
        }
    }

    if (threadIdx.x < Bc) {
        l_shared[threadIdx.x] = d_l[threadIdx.x];
        m_shared[threadIdx.x] = d_m[threadIdx.x];
    }

    float prev_m_tilda = -INFINITY;

    for (int j = 0; j < T_c; j++){
        for (int p1 = 0; p1 < head_dim; p1++){
            if (threadIdx.x < Bc) {
                v_shared[threadIdx.x * head_dim + p1] = d_v[threadIdx.x * head_dim + p1];
                k_shared[threadIdx.x * head_dim + p1] = d_k[threadIdx.x * head_dim + p1];
            }
        }

        // float m_tilda = -INFINITY;
        // float S[Bc]; 
        // float S_masked[Bc];
        prev_m_tilda = m_shared[threadIdx.x];
        for (int w1 = 0; w1 < Bc; w1++){
            float tmp = 0.0;
            int i = blockIdx.x;
            if (j < i || (j == i && w1 <= threadIdx.x)){
                for (int e1 = 0; e1 < head_dim; e1 ++){
                    tmp += q_shared[threadIdx.x * head_dim + e1] * k_shared[w1 * head_dim + e1];     
                }
                s_shared[threadIdx.x * Bc + w1] = tmp * softmax_scale;
                m_shared[threadIdx.x] = fmaxf(m_shared[threadIdx.x], s_shared[threadIdx.x * Bc + w1]);
            } else {
                s_shared[threadIdx.x * Bc + w1] = -INFINITY;
            }

        float P_tilda[Bc]; 
        float summ = 0.0;
        for (int f1 =0; f1 < Bc; f1++){
            float diff = s_shared[threadIdx.x * Bc + f1] - m_shared[threadIdx.x] ; 
            P_tilda[f1] = exp(diff); 
            summ += P_tilda[f1];
        }

        __syncthreads();

        l_shared[threadIdx.x] = exp(prev_m_tilda - m_shared[threadIdx.x]) * l_shared[threadIdx.x] + summ;

        for (int g1 = 0; g1 < head_dim; g1++){
            float first_term = 1 / (exp(prev_m_tilda - m_shared[threadIdx.x])) * o_shared[threadIdx.x * head_dim + g1];
            float tmp = 0.0;
            for (int v1 = 0; v1 < Bc; v1++){
                tmp += P_tilda[v1] * v_shared[v1 * head_dim + g1];
            }
        
            o_shared[threadIdx.x * head_dim + g1] = (first_term + tmp);
        }
        __syncthreads();
        if ((j+1) != T_c){
            d_k += Bc * head_dim;
            d_v += Bc * head_dim;
        }
        
        // d_o += Br * head_dim;
        // d_l += Br;
        // d_m += Br;
        
    }
    for (int l1 = 0; l1 < head_dim; l1++){
        if (threadIdx.x < Br) { 
            o_shared[threadIdx.x * head_dim + l1] = (1 / l_shared[threadIdx.x]) * o_shared[threadIdx.x * head_dim + l1];
            d_o[threadIdx.x * head_dim + l1] = o_shared[threadIdx.x * head_dim + l1];
        }
    }

    d_l[threadIdx.x] = m_shared[threadIdx.x] + log(l_shared[threadIdx.x]);
    }
    __syncthreads();
    
    d_o = d_o_orig;
    d_l = d_l_orig;
    d_m = d_m_orig;
}


void init_matrix(float *mat, int bs, int n_h, int s_l, int h_d){
    for (int i = 0; i < bs; i++) {
        for (int j = 0; j < n_h; j++) {
            for (int k = 0; k < s_l; k++){
                for (int l = 0; l < h_d; l++){
                    mat[i * n_h * s_l * h_d + j * s_l * h_d + k * h_d + l] = (float)rand() / RAND_MAX;
                }   
            }
        }
    }
}

double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}

int main(){
    float *h_q, *h_k, *h_v, *h_o, *h_l, *h_m;
    float *d_q, *d_k, *d_v, *d_o, *d_l, *d_m;
    float *h_S, *h_P, *h_o_cpu;
    float *d_S, *d_P, *d_o_gpu;

    size_t q_size = batch_size * n_head * sequence_length * head_dim * sizeof(float);
    size_t k_size = batch_size * n_head * sequence_length * head_dim * sizeof(float);
    size_t v_size = batch_size * n_head * sequence_length * head_dim * sizeof(float);

    size_t o_size = batch_size * n_head * sequence_length * head_dim * sizeof(float);
    size_t l_size = batch_size * n_head * sequence_length * sizeof(float);
    size_t m_size = batch_size * n_head * sequence_length * sizeof(float);

    size_t S_size = batch_size * n_head * sequence_length * sequence_length * sizeof(float);
    size_t P_size = batch_size * n_head * sequence_length * sequence_length * sizeof(float);

    h_q = (float*)malloc(q_size);
    h_k = (float*)malloc(k_size);
    h_v = (float*)malloc(v_size);

    h_o = (float*)malloc(o_size);
    h_l = (float*)malloc(l_size);
    h_m = (float*)malloc(m_size);

    h_S = (float*)malloc(S_size);
    h_P = (float*)malloc(P_size);
    h_o_cpu = (float*)malloc(o_size);

    srand(time(NULL));
    init_matrix(h_q, batch_size, n_head, sequence_length, head_dim);
    init_matrix(h_k, batch_size, n_head, sequence_length, head_dim);
    init_matrix(h_v, batch_size, n_head, sequence_length, head_dim);

    memset(h_o, 0, o_size);
    memset(h_l, 0, l_size); 

    memset(h_S, 0, S_size);
    memset(h_P, 0, P_size);

    size_t actual_m_size = m_size / sizeof(float);
    for (size_t i = 0; i < actual_m_size; i++){
        h_m[i] = -INFINITY;
    }

    CUDA_CHECK(hipMalloc(&d_q, q_size));
    CUDA_CHECK(hipMalloc(&d_k, k_size));
    CUDA_CHECK(hipMalloc(&d_v, v_size));
    CUDA_CHECK(hipMalloc(&d_o, o_size));
    CUDA_CHECK(hipMalloc(&d_l, l_size));
    CUDA_CHECK(hipMalloc(&d_m, m_size));
    CUDA_CHECK(hipMalloc(&d_S, S_size));
    CUDA_CHECK(hipMalloc(&d_P, P_size));
    CUDA_CHECK(hipMalloc(&d_o_gpu, o_size));

    CUDA_CHECK(hipMemcpy(d_q, h_q, q_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_k, h_k, k_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_v, h_v, v_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_o, h_o, o_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_l, h_l, l_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_m, h_m, m_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_S, h_S, S_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_P, h_P, P_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_o_gpu, h_o_cpu, o_size, hipMemcpyHostToDevice));

    dim3 dimGrid_mha(n_head, batch_size);
    dim3 dimBlock_mha(sequence_length);

    printf("Performing multi_attn warmup runs...\n");
    for (int i=0; i<2; i++){
        multi_head_attention<<<dimGrid_mha, dimBlock_mha>>>(d_q, d_k, d_v, d_S, d_P, d_o_gpu);
        hipDeviceSynchronize();
    }

    printf("Benchmarking multi_attn implementation...\n");
    double cpu_total_time = 0.0;
    for (int i = 0; i < 2; i++) {
        double start_time = get_time();
        multi_head_attention<<<dimGrid_mha, dimBlock_mha>>>(d_q, d_k, d_v, d_S, d_P, d_o_gpu);
        hipDeviceSynchronize();
        double end_time = get_time();
        cpu_total_time += end_time - start_time;
    }
    double cpu_avg_time = cpu_total_time / 20.0;

    CUDA_CHECK(hipMemcpy(h_o_cpu, d_o_gpu, o_size, hipMemcpyDeviceToHost));

    dim3 dimGrid_flash(T_r, batch_size * n_head);
    dim3 dimBlock_flash(Bc);

    // printf("Host: d_o address = %p\n", (void*)d_o);
    // flash_attn_kernel_2<<<dimGrid_flash, dimBlock_flash>>>(d_q, d_k, d_v, d_o, d_l, d_m);
    // CUDA_CHECK(cudaGetLastError());
    // printf("Host: d_o address after launch = %p\n", (void*)d_o);
    // cudaDeviceSynchronize();
    // printf("Host: d_o address after sync = %p\n", (void*)d_o);

    printf("Performing flash_attn warmup runs...\n");
    for (int i=0; i<2; i++){
        flash_attn_kernel_2<<<dimGrid_flash, dimBlock_flash>>>(d_q, d_k, d_v, d_o, d_l, d_m);
        hipDeviceSynchronize();
    }

    printf("Benchmarking flash_attn implementation...\n");
    double gpu_total_time = 0.0;
    for (int i = 0; i < 2; i++) {
        double start_time = get_time();
        flash_attn_kernel_2<<<dimGrid_flash, dimBlock_flash>>>(d_q, d_k, d_v, d_o, d_l, d_m);
        hipDeviceSynchronize();
        double end_time = get_time();
        gpu_total_time += end_time - start_time;
    }
    double gpu_avg_time = gpu_total_time / 20.0;

    printf("mutli-head attn avg time: %.2f milliseconds\n", cpu_avg_time * 1000);
    printf("flash attn avg time: %.2f milliseconds\n", gpu_avg_time * 1000);
    printf("Speedup: %.2fx\n", cpu_avg_time / gpu_avg_time);

    CUDA_CHECK(hipMemcpy(h_o, d_o, o_size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_l, d_l, l_size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_m, d_m, m_size, hipMemcpyDeviceToHost));

    bool correct = true;
    int mismatches = 0;
    for (int i = 0; i < batch_size; i++){
        for (int j = 0; j < n_head; j++){
            for (int k = 0; k < sequence_length; k++){
                for (int l = 0; l < head_dim; l++){
                    int index = (i * (n_head * sequence_length * head_dim)) + (j * (sequence_length * head_dim)) + (k * (head_dim)) + l;
                    if (fabs(h_o_cpu[index] - h_o[index]) > 1e-3) {
                        printf("Mismatch at [%d][%d]: multi_attn=%.6f, flash_attn=%.6f\n", i, j, h_o_cpu[index], h_o[index]);
                        mismatches++;
                        correct = false;
                    }
                    
                }
            }
        }
    }

    if (correct) {
        printf("Results are correct!\n");
    } else {
        printf("Results are incorrect (%d mismatches shown)\n", mismatches);
    }
    

    free(h_q);
    free(h_k);
    free(h_v);
    free(h_o);
    free(h_l);
    free(h_m);
    free(h_S);
    free(h_P);
    free(h_o_cpu);
    CUDA_CHECK(hipFree(d_q));
    CUDA_CHECK(hipFree(d_k));
    CUDA_CHECK(hipFree(d_v));
    CUDA_CHECK(hipFree(d_o));
    CUDA_CHECK(hipFree(d_l));
    CUDA_CHECK(hipFree(d_m));
    CUDA_CHECK(hipFree(d_S));
    CUDA_CHECK(hipFree(d_P));
    CUDA_CHECK(hipFree(d_o_gpu));

    return 0;
}

